#include "hip/hip_runtime.h"
#include "particlesystem.h"
using namespace std;


struct par{
	float3 cellSize;
	int3 gridSize;	// if grid size is same as grouping distance, then only particles in same or neighnouring cells will qualify for union
}


// Get the 3D index of the cell that contains the particle
__device__ int3 getIndex(float3 pos, float3 origin){
	int3 cell;
	cell.x = (pos.x - origin.x)/par.cellSize.x;
	cell.y = (pos.y - origin.y)/par.cellSize.y;
	cell.z = (pos.z - origin.z)/par.cellSize.z;
	return cell;
}

// calculate the hash of a given cell
__device__ int3 cellHash(int3 cell){
	//		   iz * nx * ny                     +    iy * nx            + ix
	return cell.z*par.gridSize.x*par.gridSize.y + cell.y*par.gridSize.x + cell.x;
	// can use z-order curve as hash here rather than 1D cell-index
}

// calculate the hashes of all particles (i.e. hashes of cells containing those particles)
// store the [particle id, hash] pairs (in separate arrays, index of array gives the pair id)  
__global__ calcHash(float3 * pos, uint * hashes, uint * pids, int N){
	
	uint tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= N) return;
	
	int3 cell = getIndex(pos[tid]);
	uint hash = cellHash(cell);
	
	hashes[tid] = hash;
	pids[tid]   = tid;
	
}

ParticleSystem::generateRandomClusters(float xmin, float xmax, float ymin, float ymax, float R){

}
