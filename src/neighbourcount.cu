#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <algorithm>
#include "../headers/hashtable.h"
#include "../headers/pointcloud.h"
using namespace std;

#include "../utils/simple_timer.h"
#include "../utils/cuda_vector_math.cuh"

float3 *points_dev;
int* pt_ids_dev;
HashNode* ht_dev;
int* nb_counts_dev;

__device__ void countNeighbours_hash_gpu(int3 c1, int3 c2, float Rg, int ht_size, float3*points_dev, int*pt_ids_dev, HashNode* ht_dev, int* nb_counts_dev){
	
	int count = 0;
	
//	int3 result; // (yes/no, point_id1, point_id2)
//	result.x = result.y = result.z = 0;
	
	int attempts1, attempts2;
	int id_c1 = hash_find(c1, ht_dev, ht_size, &attempts1);
	int id_c2 = hash_find(c2, ht_dev, ht_size, &attempts2);
	if (id_c1 == -1 || id_c2 == -1) return; // if either cell doesnt exist, cant merge cells
	
	for (int p1 = ht_dev[id_c1].value.x; p1 <= ht_dev[id_c1].value.y; ++p1){	
		for (int p2 = ht_dev[id_c2].value.x; p2 <= ht_dev[id_c2].value.y; ++p2){
			++count;
			
			int ip1 = pt_ids_dev[p1];
			int ip2 = pt_ids_dev[p2];
			
			float dx = points_dev[ip1].x - points_dev[ip2].x;
			float dy = points_dev[ip1].y - points_dev[ip2].y;
			float dz = points_dev[ip1].z - points_dev[ip2].z;
			
			float dist =  sqrt(dx*dx + dy*dy + dz*dz);

			if (dist < Rg){
				atomicAdd(&nb_counts_dev[ip1],1);
				atomicAdd(&nb_counts_dev[ip2],1);
			} 
		}
	}
//	cout << "attempts1 = " << attempts1 << " " << "attempts2 = " << attempts2 << endl;
	return; //result;
}


__global__ void countNeighbours_kernel(float Rd, float3*points_dev, int*pt_ids_dev, HashNode* ht_dev, int* nb_counts_dev, int ht_size, int npts, Grid par){
	
	int tid = blockIdx.x*blockDim.x + threadIdx.x;
	if (tid >= ht_size) return;

	int3 cell = ht_dev[tid].key;

	if (cell == HashNode().key) return;
	
//	int count=0;
	
	for (int ix = -2; ix <=2; ++ix){
		for (int iy = -2; iy <=2; ++iy){
			for (int iz = -2; iz <=2; ++iz){
				
				int3 cell_new;
				cell_new.x = clamp(cell.x + ix, 0, par.gridSize.x-1);
				cell_new.y = clamp(cell.y + iy, 0, par.gridSize.y-1);
				cell_new.z = clamp(cell.z + iz, 0, par.gridSize.z-1);

				if (par.cell2index(cell_new) <= par.cell2index(cell)){	
					countNeighbours_hash_gpu(cell, cell_new, Rd, ht_size, points_dev, pt_ids_dev, ht_dev, nb_counts_dev);
				}
			}
		}
	}
	

}


void PointCloud::countNeighbours_hash_gpu(float Rd){

	SimpleTimer T; T.reset(); T.start();
	Grid par;
	calcGridParams(Rd/sqrt(3), par);
	T.stop(); T.printTime("calcGrid");

	
	float3 * pos = (float3*)points.data();

	T.reset(); T.start();	
	vector <int> point_hashes(nverts);
	vector <int> point_ids(nverts);
	// get the cell ID for each particle
	for (int i=0; i<nverts; ++i) {
		int3 cell_id = par.pos2cell(pos[i]);
		point_hashes[i] = par.cell2index(cell_id);
		point_ids[i]    = i;
	}
	T.stop(); T.printTime("cell Ids");


	// sort particles by cell ID
	T.reset(); T.start();	
	sort(point_ids.begin(), point_ids.end(), [&point_hashes](int i, int j){return point_hashes[i] < point_hashes[j];}); 
	T.stop(); T.printTime("sort");


	int hashTable_size = 6000011;
	vector <HashNode> hashTable(hashTable_size);

	int n_attempts = 0;	
	int avg_attempts = 0;
	int max_attempts = 0;

	// Build Hashtable
	T.reset(); T.start();
	// start and end of each cell (both reflect indices in sorted hashes list*)
	//  -- *sorted hashes list is accessed as point_hashes[point_ids[0:nverts]]
	int start = 0, next=1;
	int cellStart = point_ids[start];
	while (next < nverts){
		int cellNext = point_ids[next];
		if (point_hashes[cellNext] != point_hashes[cellStart]){
			int2 se; se.x = start; se.y = next-1;
			
			int3 cell;
			cell.z = int(point_hashes[cellStart]/par.gridSize.x/par.gridSize.y);
			cell.y = int((point_hashes[cellStart] - cell.z*par.gridSize.x*par.gridSize.y)/par.gridSize.x);
			cell.x = int((point_hashes[cellStart] - cell.z*par.gridSize.x*par.gridSize.y - cell.y*par.gridSize.x));

			int a = hash_insert(cell, se, hashTable.data(), hashTable_size);
			++n_attempts;
			avg_attempts += a;
			max_attempts = max(max_attempts, a);

			start = next;
			cellStart = cellNext; 
		}
		else {
		}
		++next;
	}
	cout << "Insertion Attempts: " << float(avg_attempts)/n_attempts << " " << max_attempts << endl;
	T.stop(); T.printTime("map cells");

	hipMalloc(&ht_dev, hashTable_size*sizeof(HashNode));
	hipMemcpy(ht_dev, hashTable.data(), hashTable_size*sizeof(HashNode), hipMemcpyHostToDevice);
	
	hipMalloc(&points_dev, 3*nverts*sizeof(float));
	hipMemcpy(points_dev, points.data(), 3*nverts*sizeof(float), hipMemcpyHostToDevice);

	hipMalloc(&pt_ids_dev, nverts*sizeof(int));
	hipMemcpy(pt_ids_dev, point_ids.data(), nverts*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&nb_counts_dev, nverts*sizeof(int));


	neighbourCounts.resize(nverts,0);

//	particles_compared_per_cellpair = 0;
//	count_pcpc = 0;

//	// group grid cells
	int n_filled_cells = 0;
//	long long int pairs = 0;
	T.reset(); T.start();

	int nthreads = 256;
	int nblocks = (nverts-1)/nthreads+1;
	countNeighbours_kernel <<< nblocks, nthreads>>> (Rd, points_dev, pt_ids_dev, ht_dev, nb_counts_dev, hashTable_size, nverts, par);

	hipMemcpy(neighbourCounts.data(), nb_counts_dev, nverts*sizeof(int), hipMemcpyDeviceToHost);
	
//	for (int i=0; i<hashTable_size; ++i){
//		
//		int3 cell = hashTable[i].key;
//		if (compare(cell, HashNode().key)) continue;
//		++n_filled_cells;
//		
//		int count=0;
//		
//		for (int ix = -2; ix <=2; ++ix){
//			for (int iy = -2; iy <=2; ++iy){
//				for (int iz = -2; iz <=2; ++iz){
//					
//					int3 cell_new;
//					cell_new.x = clamp(cell.x + ix, 0, par.gridSize.x-1);
//					cell_new.y = clamp(cell.y + iy, 0, par.gridSize.y-1);
//					cell_new.z = clamp(cell.z + iz, 0, par.gridSize.z-1);
////					cout << "\tcell: " << cell_new.x << ", " << cell_new.y << ", " << cell_new.z << endl;

//					if (cellHash(cell_new) <= cellHash(cell)){	
////						int3 res = mergeCells(c1, c2, Rg, filled_cells, point_ids);
//						countNeighbours_hash(cell, cell_new, Rd, hashTable.data(), point_ids, hashTable_size, neighbourCounts);

//						++count;
//						++pairs;
//					}
////					if (pairs % 10000 == 0) cout << pairs << "pairs compared.\n"; 
//				}
//			}
//		}
//		
////		cells_compared_per_cell += count;
////		count_ccpc += 1;
//	}
//	cout << "pairs = " << pairs << endl;
	T.stop(); T.printTime("group cells");
	
	cout << "Summary: \n";
////	cout << "Particle pairs compared per mergeCell: " <<  float(particles_compared_per_cellpair)/count_pcpc << endl;
////	cout << "Cells compared per cell: " <<  float(cells_compared_per_cell)/count_ccpc << endl;
	cout << "No. of filled cells = " << n_filled_cells << endl;
	


}

